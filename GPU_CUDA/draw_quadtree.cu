#include "hip/hip_runtime.h"
#include"bandelet.h"

void draw_rectangle( IplImage *image, CvPoint start, CvPoint end );
void draw_line( IplImage *image, CvPoint start, CvPoint end );

void draw_quadtree(IplImage *image, int *quadtree, float *theta, int width, int j_min, int step){
	int i,j,k;
	CvPoint line_start, line_end;
	CvPoint rect_start, rect_end;
	CvPoint center;
	int coord_x, coord_y;		//coordinate of x y
	int square_wide;
	int nbr_min_side;			//the number of minmum square in a direction
	float theta_value;

	for(k = log2f(width); k >= j_min; k--){
		square_wide = power2(k);			//the width of the region that will be draw
		nbr_min_side = width / square_wide; 
		for(i = 0; i < nbr_min_side; i++)		//y direction
			for(j = 0; j < nbr_min_side; j++){	//x direction
				/*this is a leaf, transform it*/
				if(quadtree[i*square_wide*step + j*square_wide] == k){
					coord_x = j * square_wide;
					coord_y = i * square_wide;
					rect_start.x = coord_x;
					rect_start.y = coord_y;
					rect_end.x = coord_x + square_wide;
					rect_end.y = coord_y + square_wide;
					draw_rectangle(image, rect_start, rect_end );
					theta_value = theta[i*square_wide*step + j*square_wide];
					center.x = coord_x + square_wide / 2;
					center.y = coord_y + square_wide / 2;
					line_start.x = center.x - (square_wide / 2) * cos(theta_value);
					line_start.y = center.y + (square_wide / 2) * sin(theta_value);
					line_end.x = center.x + (square_wide / 2) * cos(theta_value);
					line_end.y = center.y - (square_wide / 2) * sin(theta_value);
					draw_line(image, line_start, line_end );
				}
			}
	}
	cvNamedWindow("Display Image",CV_WINDOW_AUTOSIZE);
	cvShowImage("Display Image",image);
	while( 1 ) { if( cvWaitKey( 1000 ) == 27 ) break; }
}

void draw_line( IplImage *image, CvPoint start, CvPoint end ){
	int thickness = 1;
	int lineType = 8;
	CvScalar color = {{255}};
	cvLine( image,			//line is displayed in the image 
			start,		//start point, vertical is y coordinate, horizontal is x coordinat
			end,		//end point
			color,		//RGB values of line color
			thickness,				//line thickness
			lineType,				//line is a 8-connected one
			0);
}

void draw_rectangle( IplImage *image, CvPoint start, CvPoint end ){
	int thickness = 1;
	int lineType = 8;
	CvScalar color = {{60}};
	cvRectangle(image,
			start,	//Two opposite vertices of the rectangle
			end,
			color,		//The color of the rectangle
			thickness,					//the thickness value, if is -1, the rectangle will be filled
			lineType,
			0);
}
