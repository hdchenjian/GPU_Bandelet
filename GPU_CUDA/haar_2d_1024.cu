#include "hip/hip_runtime.h"
#include"bandelet.h"

/*perform haar wavelet transform using gpu, not consider rows > 512 condition,
  for image's row > 512, we can implemetn haar transform by not using share memory,
  just using global memory, but that is slow, see another_version_haar for detail.
  a threads block computing a row of image, and a thread computing a pixel of that row
  */

__global__ void haar_2d_rows(float *image_dev, int width, int step);
__global__ void haar_2d_cols(float *image_dev, int height, int step);

void haar_2d(float *image_dev, int step, int rows, int cols, int scale, float *cost_time){
	int i;
	printf("the haar 2d transform scale is %d\n", scale);
	/*prepare data for gpu computing, do not consider rows > 512 condition*/
	int block_num, thread_num;
	int width = cols;
	int height = rows;
	block_num = rows;		/*every block compute a row or colum*/
	/*every thread load a pixel of a row, only half of thread is used to compute*/
	thread_num = cols;		
	if(1024 == cols)
		thread_num = 512;


	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	for(i = scale; i > 0; i--){
		//printf("performing haar transform scale is	%d\n", scale + 1 - i);
		haar_2d_rows<<<block_num, thread_num, thread_num * 4>>>(image_dev,width,step);
		haar_2d_cols<<<block_num, thread_num, thread_num * 4>>>(image_dev,height,step);
		block_num /= 2;
		thread_num /= 2;
		width /= 2;
		height /= 2;
	}
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	/*计算两次事件之间相差的时间（以毫秒为单位，精度为0.5微秒)*/
	hipEventElapsedTime(cost_time,start,stop);
	printf("the cost_time of haar 2d transform is %f\n", *cost_time);

	//hipHostFree(image);
	//hipFree(image_dev);
}

/*width define the region need to compute,they are change*/
/*perform the row haar transform*/
__global__ void haar_2d_rows(float *image_dev, int width, int step){
	extern __shared__ float cache[];

	int i;
	//int size = width / 2;		/*the width of next scale coarse coefficient*/
	float cache_low, cache_high;
	float coarse_coeff, detail_coeff;	/*the temporal coarse and detail coefficient*/

	for(i = 0; threadIdx.x + i * blockDim.x < width; i++){
		cache[threadIdx.x + i * blockDim.x] = image_dev[blockIdx.x * step + threadIdx.x + i * blockDim.x];
	}
	__syncthreads();		/*synchronize the threads, to get correct data*/
	
	/*we can save cache[2 * threadIdx.x] in register variable*/
	i = width / 2;
	if(threadIdx.x < i){
		cache_low = cache[2 * threadIdx.x];
		cache_high = cache[2 * threadIdx.x + 1];
		coarse_coeff = (cache_low + cache_high ) / SQRT2;
		detail_coeff = (cache_low - cache_high  ) / SQRT2;
		image_dev[blockIdx.x * step + threadIdx.x] = coarse_coeff;
		image_dev[blockIdx.x * step + threadIdx.x + i] = detail_coeff;
	}
}

/*height define the region need to compute,they are change*/
/*perform the colum haar transform*/
__global__ void haar_2d_cols(float *image_dev, int height, int step){
	extern __shared__ float cache[];

	int i;
	//int size = height / 2;		/*the width of next scale coarse coefficient*/
	float cache_low, cache_high;
	float coarse_coeff, detail_coeff;	/*the temporal coarse and detail coefficient*/

	for(i = 0; threadIdx.x + i * blockDim.x < height; i++){
		cache[threadIdx.x + i * blockDim.x] = image_dev[blockIdx.x * step + threadIdx.x + i * blockDim.x];
	}

	__syncthreads();		/*synchronize the threads, to get correct data*/
	
	i = height / 2;
	/*we can save cache[2 * threadIdx.x] in register variable*/
	if(threadIdx.x < i){
		cache_low = cache[2 * threadIdx.x];
		cache_high = cache[2 * threadIdx.x + 1];
		coarse_coeff = (cache_low + cache_high ) / SQRT2;
		detail_coeff = (cache_low - cache_high  ) / SQRT2;
		image_dev[blockIdx.x + threadIdx.x * step] = coarse_coeff;
		image_dev[blockIdx.x + threadIdx.x * step + i * step] = detail_coeff;
	}
}
