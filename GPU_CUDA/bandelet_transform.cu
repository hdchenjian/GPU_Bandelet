#include "hip/hip_runtime.h"
#include"bandelet.h"

int main(void){

	int i, j, k;			//use k to stand for scale
	//int total_nbr_bits_code;
	float elapsed_time = 0.0F;
	//int total_nbr_bits_code = 0;		 //the number of bits need to code 
	//int square_wide;
	//as src_image->imageData is char type,so need to convert it
	int rows, cols;		//it should be a square image	
	int step;
	rows = cols = HEIGHT;
	int width = cols;
	step = cols;
	
	int Jmax = log2((float)cols) -1;		//the range of scale
	int Jmin = JMIN;			//the smallest size of square(width = w^2) of wavelet transform
	int scale = log2((float)cols) - Jmin;		//the scale of haar transform			
	int s = 2;		//super-resolution factor	
	float threshold = 10.0F;  //%the threshold to evalue the best direction
	int j_min = JMIN;			//%the minimum scale for quadtree segment
	int j_max = JMIN;			//%the maximum scale for quadtree segment
	float *image;
	int *quadtree_seg;
	float *theta;
	float *thetap, *thetap_dev;			//the array that contain sampling direction
	int *map_index, *map_index_dev;		/*map_index is a 2D array map_index[length][min_side * min_side]*/
	
	int min_side = power2(j_min);		//the length of size of the min square
	int length = 2 * power2(j_min) * s + 1;	/*the number of theta value*/
	
	hipHostAlloc((void **)&image, rows * cols *sizeof(float), hipHostMallocDefault);
	hipHostAlloc((void **)&quadtree_seg, width * width * sizeof(int), hipHostMallocDefault);	
	hipHostAlloc((void **)&theta, width * width * sizeof(float), hipHostMallocDefault);	
	hipHostAlloc((void **)&map_index,length*min_side*min_side*sizeof(int),hipHostMallocDefault);		
	hipHostAlloc((void **)&thetap, length * sizeof(float), hipHostMallocDefault);

	float adder = PI / (length - 1);	
	for(i = 0; i < length - 1; i++){
		thetap[i] = (adder / 2) + i * adder;
	}
	thetap[i] = NO_GEO;		//add no geometry stream direction
		
	float theta_value;
	float x_direct, y_direct;
	float *grid = (float *)malloc(min_side * min_side * sizeof(float));
	float *gridp = grid;
	float *grid_sort = (float *)malloc(min_side * min_side * sizeof(float));
	/*we need to keep grid,because we need free it,so we can not change it*/	
	
	for(i = 0; i < length; i++){
		theta_value = thetap[i];
		x_direct = -sinf(theta_value);
		y_direct = cosf(theta_value);
		/*projection on orthogonal direction*/
		/*this grid was created using matlab colums order, not C's row order, be careful,
		 * x coordinate is vertical, and y coordinate is horizontal, as an image coordinate*/

		for(j=1;j<min_side+1;j++)			//y_direct
			for(k=1;k<min_side+1;k++){ 	//x_direct
				*gridp = x_direct * k + y_direct * j;
				gridp++;
			}
		gridp = grid;
		sort(gridp,grid_sort,min_side * min_side);	/*sort points in increasing order*/
		/*map gridp to grid_sort, get the index of sorting wavelet coefficient*/
		mapping(grid_sort,gridp,map_index + i * min_side * min_side,min_side * min_side); //have a bug
	}
	
	FILE *image_matrix;
	if ((image_matrix = fopen("image_matrix", "r+")) != NULL){
		i = fread(image, sizeof(float), rows * cols, image_matrix);
		printf("the number of element of fread is %d\n", i);
		fclose(image_matrix);
	}
	else
		printf("fail opening the file\n");	

	printf("get the processed image\n");
	printf("rows is %d, cols is %d\n",rows, cols);
	printf("image row step in bytes is %d, be careful to use it \n", step);
	

	/*initial the theta to NO_GEO*/	
	for(i=0;i<rows;i++)
		for(j=0;j<cols;j++)
			*(theta + i*step + j) = NO_GEO;
	/*initial the quadtree_seg to j_min*/	
	for(i=0;i<rows;i++)
		for(j=0;j<cols;j++)
			*(quadtree_seg + i*step + j) = j_min;

	/*prepare data for gpu computing, do not consider rows > 512 condition*/
	float *image_dev;
	int *quadtree_seg_dev;
	float *theta_dev;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	hipMalloc((void **)&theta_dev, rows * cols * sizeof(float));
	hipMalloc((void **)&image_dev, rows * cols * sizeof(float));
	hipMalloc((void **)&quadtree_seg_dev, rows * cols * sizeof(int));
	hipMalloc((void **)&thetap_dev, length * sizeof(float));
	hipMalloc((void **)&map_index_dev, length * min_side * min_side * sizeof(int));
	
	hipMemcpy(image_dev, image, rows * cols * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(quadtree_seg_dev, quadtree_seg, rows *cols* sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(theta_dev, theta, rows * cols * sizeof(float),  hipMemcpyHostToDevice);
	hipMemcpy(thetap_dev, thetap, length  * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(map_index_dev, map_index, length* min_side* min_side * sizeof(int), hipMemcpyHostToDevice);
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("copy data to GPU consume time is %f ms\n", elapsed_time);
	
	float time_temp;
	haar_2d(image_dev, step, rows, cols, scale, &time_temp);
	elapsed_time += time_temp;
	printf("2D haar transform over\n");
	
	compute_quadtree(image_dev, width, step, threshold, j_min, j_max, s, quadtree_seg_dev,\
			theta_dev, &time_temp, thetap_dev, map_index_dev);
	elapsed_time += time_temp;
	//printf("compute_quadtree consume time is %f ms\n", elapsed_time);
	
	printf("compute_quadtree over \n");
			
	/*perform bandelet transform for each square*/
	time_temp = 0.0F;
	//perform_bandelet_transform(image, j_min, image_dev, width, step, quadtree_seg_dev, theta_dev, &total_nbr_bits_code, &time_temp, length, thetap_dev, map_index_dev);
	//elapsed_time += time_temp;
	//printf("the total_nbr_bits_code is %d\n",total_nbr_bits_code);
	hipEventRecord(start, 0);
	hipMemcpy(quadtree_seg, quadtree_seg_dev, rows * cols*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(theta, theta_dev, rows * cols * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(image, image_dev, rows * cols * sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_temp, start, stop);
	printf("copy back data to CPU consume time is %f ms\n", time_temp);
	elapsed_time += time_temp;
	printf("the total elapsed_time is %f ms\n", elapsed_time);
	
	/*save the result as binary file*/
	int volume = rows * cols;
	FILE *quadtree_result;
	FILE *theta_result;
	FILE *bandelet_coefficient;
	if ((quadtree_result = fopen("result/quadtree_result", "w+")) != NULL){
		fwrite(quadtree_seg, sizeof(int), volume, quadtree_result);
		fclose(quadtree_result);
	}
	else
		printf("fail opening the file\n");

	if ((theta_result = fopen("result/theta_result", "w+")) != NULL){
		fwrite(theta, sizeof(float), volume, theta_result);
		fclose(theta_result);
	}
	else
		printf("fail opening the file\n");

	if ((bandelet_coefficient = fopen("result/bandelet_coefficient", "w+")) != NULL){
		fwrite(image, sizeof(float), volume, bandelet_coefficient);
		fclose(bandelet_coefficient);
	}
	else
		printf("fail opening the file\n");

	printf("draw the quadtree segment\n");
	printf("the width is %d, the j_min is %d, the step is %d\n", cols, j_min, step);
	/*adjust the theta value, so plot the correct geometry stream*/
	for(i = 0; i < rows; i++)
		for(j = 0; j < cols; j++){
			if(theta[i * step + j] < PI /2)
				theta[i * step + j] = PI / 2 + theta[i * step + j];
			else
				theta[i * step + j] = theta[i * step + j] - PI / 2;
		}
	
	//draw_quadtree(src_image, quadtree_seg, theta, cols, j_min, step);
	
	free(grid);
	free(grid_sort);
	
	hipHostFree(thetap);
	hipHostFree(map_index);
	hipHostFree(image);
	hipHostFree(quadtree_seg);
	hipHostFree(theta);

	hipFree(thetap_dev);
	hipFree(map_index_dev);
	hipFree(image_dev);
	hipFree(quadtree_seg_dev);
	hipFree(theta_dev);
	//hipFree(theta_next_scale_dev);
	//cvReleaseImage(&src_image);

	return 0;
}
