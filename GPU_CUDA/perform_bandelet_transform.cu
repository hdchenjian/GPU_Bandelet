/*perform bandelet transform for a region*/

#include"bandelet.h"

void perform_bandelet_transform(float *image, int j_min, float *image_dev, int width, int step, \
		int *quadtree_seg_dev, float *theta_dev, int *total_bits, float *time_transform, int length, float *thetap_dev, int *map_index_dev){

	int square_wide;
	int nbr_square;
	int *total_bits_dev;

	dim3 grid, block;
	square_wide = power2(j_min);
	nbr_square = width / square_wide;
	grid.x = nbr_square;
	grid.y = nbr_square;
	block.x = square_wide * square_wide;
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	//hipMalloc((void **)&total_bits_dev, sizeof(int));
	//hipMemset(total_bits_dev, 0, sizeof(int));
  
	transform<<<grid, block, square_wide * square_wide * 4 * 4>>>(step, square_wide, image_dev,  theta_dev, quadtree_seg_dev, j_min, length, thetap_dev, map_index_dev);

	//hipMemcpy(total_bits, total_bits_dev, sizeof(int), hipMemcpyDeviceToHost);
	//printf("the total_nbr_bits_code is %d\n", *total_bits);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(time_transform, start, stop);
	printf("perform_bandelet_transform consume time is %f ms\n", *time_transform);

}
