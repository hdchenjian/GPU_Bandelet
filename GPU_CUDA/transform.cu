#include "hip/hip_runtime.h"
/*perform bandeletization for a small square, width is the width of current scale 
  *square, this have a difference using of share memory, cache's last
  *4byte(cache[region_element_num * 4]) is used for distingushing
  *whether have same element in gridp_sort, be careful*/

#include"bandelet.h"

__global__ void transform(int step, int width, float *image_dev, float *theta_dev, int *quadtree_seg_dev, int scale, int length, float *thetap_dev, int *map_index_dev){

	extern __shared__ float cache[];

	int j, k;
	float theta_value;
	int region_element_num = width * width;
	float *region = image_dev + blockIdx.y * width * step + blockIdx.x * width;
	float *region_data = cache;		//we can save this share mem by using a register variable
	float *gridp = region_data + region_element_num;
	float *gridp_sort = region_data + 2 * region_element_num;
	int *map = (int *)(region_data + 3 * region_element_num);

	if(0 == threadIdx.x){		//note:can not use temp, because only thread 1 's temp is change
		cache[1] = 0.0F;		//a sign bit that mean whether perform bandeletization
		map[0] = 0;				//a sign bit that mean whether have same element in gridp_sort
		cache[0] = theta_dev[blockIdx.y * width * step + blockIdx.x * width];
		if(cache[0] != NO_GEO){
			cache[1] = 1.0F;
		}
	}
	__syncthreads();		/*synchronize the threads, to get correct data*/

	if(1.0F == cache[1]){		//need to perform bandeletization
		theta_value = cache[0];
	}
	else						//square scale < current scale or theta_value == NO_GEO
		return ;

	/*load the region data to share memory */
	region_data[threadIdx.x] = region[threadIdx.x / width * step + threadIdx.x % width];
	__syncthreads();		/*synchronize the threads, to get correct data*/
	
	for(j = 0; j < length; j++){
			if(theta_value == thetap_dev[j]){
				break;
			}
		}
		
	k = (map_index_dev + j * blockDim.x)[threadIdx.x];
	//map[threadIdx.x] = (map_index_dev + j * blockDim.x)[threadIdx.x];

	/*mapping region data to a sequence based on map_index, use gridp store mapping data*/
	//for every colums, using colums order
	//have a bug, changed
	gridp[threadIdx.x] = region_data[k % width * width + k / width];
	__syncthreads();		/*synchronize the threads, to get correct data*/

	/*perform haar 1D transform, gridp is input data, use gridp_sort store result*/
	j = region_element_num;		/*the width of next scale coarse coefficient*/
	while(j > 1){
		j /= 2;
		if(threadIdx.x < j){
			gridp_sort[threadIdx.x] = 
				(gridp[2 * threadIdx.x] + gridp[2 * threadIdx.x + 1 ]) / SQRT2;
			gridp_sort[threadIdx.x + j] = 
				(gridp[2 * threadIdx.x] - gridp[2 * threadIdx.x + 1 ]) / SQRT2;
		}
		__syncthreads();		/*synchronize the threads, to get correct data*/
		if(threadIdx.x < j)		/*prepare data for next scale wavelet transform*/
			gridp[threadIdx.x] = gridp_sort[threadIdx.x];
		__syncthreads();		/*synchronize the threads, to get correct data*/
	}

	/* map haar coefficient based map index*/
	*(region + (k % width) * step + k / width) =
													gridp_sort[threadIdx.x];
}
