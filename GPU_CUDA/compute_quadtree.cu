#include "hip/hip_runtime.h"
/* compute the quadtree that optimize the Lagrangian
 * j_min is the depth minimum of the QT,
		 * (ie 2^j_min is the size minimum of the square).
 * j_max is the depth maximum of the QT        [default : min(5,log2(n))]
 * (ie 2^j_max is the size maximum of the square).
 * s is the super-resolution for the geometry [default 2]
 * quadtree_seg and theta is the output
 * quadtree_seg is an image representing the levels of the quadtree.
 * theta is an image representing the optimal angle choosed on each
 */

#include"bandelet.h"

void compute_quadtree(float *image_dev, int width, int step, float threshold, int j_min,\
		int j_max, int s, int *quadtree_seg_dev, float *theta_dev, float *time_quadtree, float *thetap_dev, int *map_index_dev){

	int min_side = power2(j_min);		//the length of size of the min square
	/*sample (2 * power(j_min) * s) theta value, so a thread block have (2 * power(j_min) * s)
	 * threads, and we can not sample more than power(j_min)^2 direction, because when
	 * j_min=5, length=1024 > 512, and length should be multiplier of 32*/
	int length = 2 * power2(j_min) * s + 1;	/*the number of theta value*/	
					
	dim3 grid_num(width / min_side, width / min_side, 1);
	dim3 block(min_side * min_side, 1, 1);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	/* a threads block compute a square region which size is (2^j_min),and we use
	 * a for loop * compute all direction, one loop compute a direction, the 
	 * number of thread of a tread block is the number of pixel of a square region*/
	compute_best_direction<<<grid_num, block, min_side * min_side * 4 * 4 + 24>>>(threshold, step, min_side,thetap_dev, image_dev, theta_dev, length, map_index_dev);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(time_quadtree, start, stop);
	printf("compute_quadtree consume time is %f ms\n", *time_quadtree);

}
