#include "hip/hip_runtime.h"
#include"bandelet.h"

/*compute the best direction of a image square, the maximum thread of a block is 512,
 * so the width < 5, for compute capability >= 2.0 devices, the maximum thread of a 
 * block is 1024, so width < 6, and a thread need to process more than one data
 * the best_theta is the best direction,
 * the min_lag is the minimum Lagrangian, the x coordinate of a thread block is horizontal
 * the width is the length of min_side. nbr_coefficient, total_error, min_lag, and
 * best_theta parameters passed by share memory, so we can store some
 * variable in share memory, and that can save some register, initial parameters :
 * nbr_coefficient = 0, total_error = 0.0F, min_lag = 1000000.0F(a big float number)
 * best_theta = NO_GEO
 * The method for passing kernel parameters varies with architecture. Compute capability
 * 1.* devices put the values in shared memory. Compute capability >= 2.0 put the values
 * in constant memory, the share memory is min_side * min_side * 4 * 4 + 16, be careful*/

/*shared memory map cache[0]: grid_sort; cache[width *width] : grid; cache[width *width * 2] :
  region_data; cache[width *width * 3] : coefficient_save */
/*this function combine the findind direction and perform bandelet transform*/

__global__ void compute_best_direction(float threshold,int step,int width, float *thetap_dev, \
		float *image_dev, float *theta_dev, int length, int *map_index_dev){
	/*use for warp wavelet translation */
	extern __shared__ float cache[];
	
	int j, k;
	//float theta_value;
	//int region_element_num = width * width;
	float temp;
	//float *warp_wavelet_coeff;
	
	//float *region = image_dev + blockIdx.y * width * step + blockIdx.x * width;
	//float *region_data = cache;		//we can save this share mem by using a register variable
	float *gridp = cache + blockDim.x;
	//int *index_of_map = (int *)(cache  + 4 * blockDim.x + 0);
	//float *gridp_sort = cache + 2 * blockDim.x;
	//int *coefficient_save = (int *)(cache + 3 * blockDim.x);
	//int *nbr_coefficient = (int *)(cache  + 4 * blockDim.x);
	//float *total_error = (float *)((cache  + 4 * blockDim.x) + 1);
	//float *min_lag = total_error + 1;
	//float *best_theta = min_lag + 1;
	//int *mapping;

	//we do not transform the most coarse scale wavelet coefficient
	if(0 == blockIdx.x && 0 == blockIdx.y)
		return;

	if(0 == threadIdx.x){
		//*nbr_coefficient = 0;
		//*total_error = 0.0F;
		//*min_lag = 1000000.0F;

		//*((cache  + 4 * blockDim.x) + 0) = 0.5F;	//the index of thetap that save best_theta
		//*((cache  + 4 * blockDim.x) + 1)	//the number of coefficient that above threshold
		*((cache  + 4 * blockDim.x) + 2) = 1000000.0F; //error + LAMBDA * T * threshold * threshold
		*((cache  + 4 * blockDim.x) + 3) = NO_GEO;		//best_theta
		//theta_value = *((cache  + 4 * blockDim.x) + 4)	//the theta_value of current loop
	}

	/*load the region data to share memory */
	(cache + 2 * blockDim.x)[threadIdx.x] = (image_dev + blockIdx.y * width * step + blockIdx.x * width)[threadIdx.x / width * step + threadIdx.x % width];
	__syncthreads();		/*synchronize the threads, to get correct data*/

/*perform warp wavelet transform */
for(k = 0; k < length; k++){
	//mapping = map_index_dev + k * blockDim.x;
	if(0 == threadIdx.x){
		*((cache  + 4 * blockDim.x) + 4) = thetap_dev[k];	//get the theta_value of current loop
	}
	__syncthreads(); 

	if(*((cache  + 4 * blockDim.x) + 4) != NO_GEO){

		/*mapping region data to a sequence based on map_index, use gridp store mapping data*/
		//for every colums, using colums order
		//gridp[map[threadIdx.x]] = region_data[threadIdx.x / width + (threadIdx.x % width)*width];
		//have a bug, changed
		//gridp[threadIdx.x] = region_data[map[threadIdx.x] % width*step + map[threadIdx.x] / width];
		j = (map_index_dev + k * blockDim.x)[threadIdx.x];
		gridp[threadIdx.x] = (cache + 2 * blockDim.x)[j % width * width + j / width];

		__syncthreads();		/*synchronize the threads, to get correct data*/
		/*perform haar 1D transform, gridp is input data, use gridp_sort store result*/
		j = blockDim.x / 2;		/*the width of next scale coarse coefficient*/
		while(j > 1){
			if(threadIdx.x < j){
				cache[threadIdx.x] = 
					(gridp[2 * threadIdx.x] + gridp[2 * threadIdx.x + 1 ]) / SQRT2;
				cache[threadIdx.x + j] = 
					(gridp[2 * threadIdx.x] - gridp[2 * threadIdx.x + 1 ]) / SQRT2;
			}
			if(threadIdx.x < j)		/*prepare data for next scale wavelet transform*/
				gridp[threadIdx.x] = cache[threadIdx.x];
			j /= 2;
			__syncthreads();		/*synchronize the threads, to get correct data*/
		}

		//warp_wavelet_coeff = cache;
		/*perform_quantization, the input data store at gridp_sort or region_data,
		 * the output data store at gridp
		 * The quantizer is defined by y=Q_T(x) where:
		 *Q_T(x) = 0    if  |x|<T
		 *Q_T(x) = sign(x) * ([|x| / T]+0.5)*T      where [.]=floor
		 *(i.e. a nearly uniform quantizer with twice larger zero bin)*/
		temp = cache[threadIdx.x];
		if(fabsf(temp) < threshold){
			gridp[threadIdx.x] = 0.0F;
			cache[threadIdx.x] = 0.0F;
		}
		else{ 
			gridp[threadIdx.x] = ((temp < 0.0F)? (-1.0F) : (1.0F)) * 
				(floorf( fabsf(temp) / threshold ) + 0.5F) * threshold;
			cache[threadIdx.x] = 1.0F;
		}
		__syncthreads();		/*synchronize the threads, to get correct data*/

		/*compute the number of coefficient above threshold and the approximation error*/
		j = blockDim.x / 2;
		while (j != 0) {  /*reduce algorithem compute the number of coefficient above threshold*/
			if(threadIdx.x < j)
				cache[threadIdx.x] += cache[threadIdx.x + j];
			j /= 2;
			__syncthreads();
		}

		/*as we have got the value that store at gridp[threadIdx.x], so we can use gridp
		 * store the error that we cover the gridp content, be careful*/
		temp -= gridp[threadIdx.x];
		gridp[threadIdx.x] = temp * temp;		// error = temp * temp;
		__syncthreads();        /*synchronize the threads, to get correct data*/
		j = blockDim.x / 2;	//store at share memory pass by parameter
		while (j != 0) {  /*reduce algorithem compute the total error*/
			if(threadIdx.x < j)
				gridp[threadIdx.x] += gridp[threadIdx.x + j];
			j /= 2;
			__syncthreads();
		}
	}
	else{	//for no geometry stream, just copy region to region_data
		//warp_wavelet_coeff = (cache + 2 * blockDim.x);

		/*perform_quantization, the input data store at gridp_sort or region_data,
		 * the output data store at gridp
		 * The quantizer is defined by y=Q_T(x) where:
		 *Q_T(x) = 0    if  |x|<T
		 *Q_T(x) = sign(x) * ([|x| / T]+0.5)*T      where [.]=floor
		 *(i.e. a nearly uniform quantizer with twice larger zero bin)*/
		temp = (cache + 2 * blockDim.x)[threadIdx.x];
		if(fabsf(temp) < threshold){
			gridp[threadIdx.x] = 0.0F;
			cache[threadIdx.x] = 0.0F;
		}
		else{ 
			gridp[threadIdx.x] = ((temp < 0.0F)? (-1.0F) : (1.0F)) * 
				(floorf( fabsf(temp) / threshold ) + 0.5F) * threshold;
			cache[threadIdx.x] = 1.0F;
		}
		__syncthreads();		/*synchronize the threads, to get correct data*/

		/*compute the number of coefficient above threshold and the approximation error*/
		j = blockDim.x / 2;
		while (j != 0) {  /*reduce algorithem compute the number of coefficient above threshold*/
			if(threadIdx.x < j)
				cache[threadIdx.x] += cache[threadIdx.x + j];
			j /= 2;
			__syncthreads();
		}

		/*as we have got the value that store at gridp[threadIdx.x], so we can use gridp
		 * store the error that we cover the gridp content, be careful*/
		//temp = gridp[threadIdx.x] - (cache + 2 * blockDim.x)[threadIdx.x];
		temp -= gridp[threadIdx.x];
		gridp[threadIdx.x] = temp * temp;		// error = temp * temp;
		__syncthreads();        /*synchronize the threads, to get correct data*/
		j = blockDim.x / 2;	//store at share memory pass by parameter
		while (j != 0) {  /*reduce algorithem compute the total error*/
			if(threadIdx.x < j)
				gridp[threadIdx.x] += gridp[threadIdx.x + j];
			j /= 2;
			__syncthreads();
		}
	}


	/* use the first thread compute the min_lag and best_theta*/
	if(0 == threadIdx.x){
		/*do not forget the nbr_coefficient and total_error*/
		*(cache  + 4 * blockDim.x + 1) = cache[0];	//nbr_coefficient above threshold	
		if(*((cache  + 4 * blockDim.x) + 4) != NO_GEO){ 
		//*total_error = gridp[0],	temp save the Lagrangian = ERROR + LAMBDA * (Rg + Rb) * T^2
			temp = gridp[0] + LAMBDA * (*(cache  + 4 * blockDim.x + 1) * GAMMA + 1 + \
					ceilf(log2f(length))) * threshold * threshold;
		}
		else{
			temp = gridp[0] + \
				   LAMBDA * (*(cache  + 4 * blockDim.x + 1) * GAMMA + 1) * threshold * threshold;
		}
		if(*((cache  + 4 * blockDim.x) + 2) > temp){
			*((cache  + 4 * blockDim.x) + 2) = temp;
			*((cache  + 4 * blockDim.x) + 3) = *((cache  + 4 * blockDim.x) + 4);
			//save the index of thetap to get the map_index used by perform_bandelet_transform
			*(((int *)cache)  + 4 * blockDim.x ) = k;
		}
	}
}

	__syncthreads();
	*( theta_dev + (blockIdx.y * width * step + blockIdx.x *width) + //start address
		( threadIdx.x / width * step + threadIdx.x % width ) ) =	//offset address 
								*((cache  + 4 * blockDim.x) + 3);
								
	/*perform_bandelet_transform*/
	temp = *((cache  + 4 * blockDim.x) + 3);	//the best theta of this region
	if(temp == NO_GEO)
		return ;

	j = *(((int *)cache)  + 4 * blockDim.x ); //save the index of thetap to get the map_index
	k = (map_index_dev + j * blockDim.x)[threadIdx.x];	//k save the map_index of current thread
	
	//map region data to a sequence based on map_index
	gridp[threadIdx.x] = *((cache + 2 * blockDim.x) + k % width * width + k / width);
	__syncthreads();		/*synchronize the threads, to get correct data*/

	/*perform haar 1D transform, gridp is input data, use gridp_sort store result*/
	j = blockDim.x;		/*the width of next scale coarse coefficient*/
	while(j > 1){
		j /= 2;
		if(threadIdx.x < j){
			cache[threadIdx.x] = 
				(gridp[2 * threadIdx.x] + gridp[2 * threadIdx.x + 1 ]) / SQRT2;
			cache[threadIdx.x + j] = 
				(gridp[2 * threadIdx.x] - gridp[2 * threadIdx.x + 1 ]) / SQRT2;
		}
		if(threadIdx.x < j)		/*prepare data for next scale wavelet transform*/
			gridp[threadIdx.x] = cache[threadIdx.x];
		__syncthreads();		/*synchronize the threads, to get correct data*/
	}

	/* map haar coefficient based map index*/
	*(image_dev + blockIdx.y * width * step + blockIdx.x * width		//start address
			 + (k % width) * step + k / width) = cache[threadIdx.x];
}
